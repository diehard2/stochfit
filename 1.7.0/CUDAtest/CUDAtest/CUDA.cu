#include "hip/hip_runtime.h"
//defines
#include "defines.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <conio.h>
#include <float.h>
// includes, project
#include <cutil.h>
//#define CUDADEBUG
// includes, kernels
#include <kernel.cu>


extern "C" void CUDAInit(float** genomearray, float** doublednk, float** cudank, float2** nk, float* dz, float* m_dboxsize, int boxcount, int npts, float** scratcharray)
{
	//Initialization
    
    int deviceCount;                                                         
    CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));                
    CUDA_SAFE_CALL(hipSetDevice(deviceCount-1));     
  
    
    *nk = (float2*)malloc(npts*sizeof(float2));
    CUDA_SAFE_CALL(hipMalloc((void**) genomearray, (boxcount+2)*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**) doublednk, npts*sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**) cudank, npts*sizeof(float)));
     CUDA_SAFE_CALL(hipMalloc((void**) scratcharray, npts*sizeof(float)*(boxcount+2)));
    
	//Setup the arrays
	*dz = 1.0f/RESOLUTION;
	*m_dboxsize = (LAYERLENGTH+7.0f)/((float)boxcount);


    
    setarrays<<<1, 1 >>>(*genomearray, BOXCOUNT);
}

extern "C" void CUDAFreeArrays(float** genomearray, float** doublednk, float** cudank, float2** nk, float** scratcharray)
{
	//Free memory
	CUDA_SAFE_CALL(hipFree(*genomearray));
    CUDA_SAFE_CALL(hipFree(*doublednk));
	CUDA_SAFE_CALL(hipFree(*cudank));
	CUDA_SAFE_CALL(hipFree(*scratcharray));
    free(*nk);
}

extern "C" void
CUDAMakeDensity(const int argc, const char** argv, int points, float* genomearray,float* cudank, float2* nk, float* doublenk, int boxcount, bool writenk, float* scratcharray)
{
#ifndef BLANK
   unsigned int num_threads = 128;
   int boxes = BOXCOUNT + 1; 
   
   if(BOXCOUNT+2 > num_threads)
		num_threads = BOXCOUNT+2;
   
  
   int pts = TOTALLENGTH*RESOLUTION;
   float boxsize = (LAYERLENGTH+7.0f)/((float)boxcount);
   float dz = 1.0f/RESOLUTION;
 
   

   float rough = dz/(ROUGHNESS * sqrtf(2.0f));
 
   int neededmemory = boxes*sizeof(float)*3;
	
	dim3 dimBlock (num_threads,1);
    dim3 dimGrid (ceil((float)pts/(float)dimBlock.x),1);
    

   EDCalc<<<dimGrid, dimBlock, neededmemory>>> (genomearray, cudank, doublenk, boxes,rough,  RHO_A*0.5f, pts, boxsize, dz, LEFTOFFSET, scratcharray);
//	FasterEDCalc<<<dimGrid, dimBlock, neededmemory>>> (genomearray, cudank, distarray,boxes,rough, RHO_A*0.5f, pts, boxsize, dz, LEFTOFFSET);
	

    

  	if(writenk)
	{
	    float* temp = (float*)malloc(pts*sizeof(float));
		CUDA_SAFE_CALL(hipMemcpy(temp, cudank, pts*sizeof(float), hipMemcpyDeviceToHost));
		
		for(int i = 0; i < pts; i++)
		{
		  nk[i].x = temp[i];
		  
		}
		
		free(temp);	
		  
	}

#else
unsigned int num_threads = 1;
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
 
   for(int i = 0; i < ITERATIONS; i++)
   {
		blank<<<grid, threads>>>();
   }
   #endif
	
}
