#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_
#define _KERNEL_H_

extern __shared__ float combined[];

__global__ void
EDCalc( float* genome, float* dnk, float* doublednk, int refllayers, float roughness,  float rho, int totalpts, float boxsize, float dz, float leftoffset, float* scratcharray)
{
	int tid = threadIdx.x;
	int index = blockIdx.x*blockDim.x+tid;
	float *srhoarray, *sdrhoarray, *sdistarray;
	

	srhoarray = combined;
	sdrhoarray = combined + refllayers;
	sdistarray = combined + refllayers + refllayers;

	if(tid < refllayers)
	{
		srhoarray[tid] = (genome[tid+1]-genome[tid])*rho;
		sdrhoarray[tid] = 2.0f*srhoarray[tid];
		sdistarray[tid] = (-leftoffset- tid*boxsize)/dz;
	}
	
	
	if(index < totalpts)
	{
	  
		float temp = 0;
		float dist = 0;
		for(int k = 0; k < refllayers; k++)
		{
			
			dist = (index + sdistarray[k] )*roughness;
			
			if(dist > 4.0f)
				temp += sdrhoarray[k];
			else if (dist > -4.0f)
				temp += (srhoarray[k])*(1.0f+erff(dist));
			
		}
		dnk[index] = temp;
		doublednk[index] = temp+temp;
	 }
 
 __syncthreads();
 
 //Start the reflectivity calculation

}

__global__ void blank()
{

}

__global__ void setarrays(float* genomearray, int boxcount)
{
    genomearray[0] = 0.0f;
	for(int i = 1; i < boxcount+1; i++)
	{
		genomearray[i] = 1.2f;
	
	}
	genomearray[boxcount + 1] = 1.0f;
	

   syncthreads();
}


//working
__global__ void
EDCalcWorking( float* genome,float* edspacing,float* rhoarray, float2* dnk, float* distarray, int ptsperthread, int refllayers, float roughness )
{

	 const unsigned int tid = threadIdx.x;
     float2* nk = dnk + tid*ptsperthread;
     float* edspace = edspacing + tid*ptsperthread;

      float temp = 0;
      for(int i = 0; i < ptsperthread; i++)
 		{
			temp = 0;
			for(int k = 0; k < refllayers; k++)
			{
				
				float dist = (edspace[i]-distarray[k] )*roughness;
				if(dist > 6.0f)
				{
					temp += (rhoarray[k])*(2.0f);
				}
				else if (dist > -6.0f)
				{
				//	temp += (rhoarray[k])*(1.0f+erff(dist));
				}
			}
			nk[i].x = temp;
	
		}

}


//Working faster
__global__ void
FasterEDCalc( float* genome, float* dnk, float* distarray, int refllayers, float roughness,  float rho, int totalpts, float boxsize, float dz, float leftoffset)
{

int tid = threadIdx.x;
	int index = blockIdx.x*blockDim.x+tid;
	float *srhoarray, *sdrhoarray, *sdistarray;

	srhoarray = combined;
	sdrhoarray = combined + refllayers;
	sdistarray = combined + 2*refllayers;

	if(tid < refllayers)
	{
		srhoarray[tid] = (genome[tid+1]-genome[tid])*rho;
		sdrhoarray[tid] = 2.0f*srhoarray[tid];
		sdistarray[tid] = (-leftoffset- tid*boxsize)/dz;
	}
	
	
if(index < totalpts)
{
  
    float temp = 0;
    float dist = 0;
	for(int k = 0; k < refllayers; k++)
	{
		
		dist = (index + sdistarray[k] )*roughness;
		
		if(dist > 4.0f)
			temp += sdrhoarray[k];
		else if (dist > -4.0f)
			temp += (srhoarray[k])*(1.0f+erff(dist));
		
	}
	dnk[index] = temp;

 }
 
 __syncthreads();


 
}



#endif //_KERNEL_H_

